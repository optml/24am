/*
 *
 * This is a parallel sparse PCA solver
 *
 * The solver is based on a simple alternating maximization (AM) subroutine
 * and is based on the paper
 *    P. Richtarik, M. Takac and S. Damla Ahipasaoglu
 *    "Alternating Maximization: Unified Framework and 24 Parallel Codes for L1 and L2 based Sparse PCA"
 *
 * The code is available at https://code.google.com/p/24am/
 * under GNU GPL v3 License
 *
 */

#include <stdio.h>
#include <stdlib.h>

#include "../gpower/sparse_PCA_solver.h"
#include "../utils/file_reader.h"
#include "../utils/option_console_parser.h"
#include "../gpugpower/gpu_sparse_PCA_solver.h"
using namespace SolverStructures;
#include "../utils/file_reader.h"
#include "../utils/option_console_parser.h"
#include "experiment_utils.h"
#include "../problem_generators/gpower_problem_generator.h"

template<typename F>
int test_solver(SolverStructures::OptimizationSettings * optimizationSettings) {
	SolverStructures::OptimizationStatisticsistics* optimizationStatistics =
			new OptimizationStatisticsistics();

	ofstream fileOut;
	fileOut.open("results/paper_experiment_gpu_speedup.txt");
	ofstream fileOutCPU;
	fileOutCPU.open("results/paper_experiment_gpu_speedup_cpu.txt");

	cublasoptimizationStatisticsus_t optimizationStatisticsus;
	hipblasHandle_t handle;
	optimizationStatisticsus = hipblasCreate(&handle);
	if (optimizationStatisticsus != CUBLAS_optimizationStatisticsUS_SUCCESS) {
		fprintf(stderr, "! CUBLAS initialization error\n");
		return EXIT_FAILURE;
	} else {
		printf("CUBLAS initialized.\n");
	}

	hipDeviceProp_t dp;
	hipGetDeviceProperties(&dp, 0);
	optimizationSettings->gpu_sm_count = dp.multiProcessorCount;
	optimizationSettings->gpu_max_threads = dp.maxThreadsPerBlock;

	mytimer* mt = new mytimer();
	std::vector<F> B_mat;
	std::vector<F> y;
	int multSC = 1;
	for (int mult = multSC; mult <= 64; mult = mult * 2) {
		int m = 100 * mult;
		int n = 1000 * mult;

		const int MEMORY_BANK_FLOAT_SIZE = MEMORY_ALIGNMENT / sizeof(F);
		const unsigned int LD_M = (
				m % MEMORY_BANK_FLOAT_SIZE == 0 ?
						m :
						(m / MEMORY_BANK_FLOAT_SIZE + 1)
								* MEMORY_BANK_FLOAT_SIZE);
		const unsigned int LD_N = (
				n % MEMORY_BANK_FLOAT_SIZE == 0 ?
						n :
						(n / MEMORY_BANK_FLOAT_SIZE + 1)
								* MEMORY_BANK_FLOAT_SIZE);
		thrust::host_vector<F> h_B(LD_M * n, 0);
		generateProblem(n, m, &h_B[0], m, n, false);
		optimizationSettings->max_it = 100;
		optimizationSettings->toll = 0;
		optimizationSettings->penalty = 0.02;
		optimizationSettings->constrain = n / 100;
		optimizationSettings->algorithm = L1_penalized_L1_PCA;
		optimizationSettings->onTheFlyMethod = false;
		optimizationSettings->gpu_use_k_selection_algorithm = false;
		optimizationStatistics->n = n;
		// move data to DEVICE
		thrust::device_vector<F> d_B = h_B;
		// allocate vector for solution
		thrust::host_vector<F> h_x(n, 0);

		for (optimizationSettings->starting_points = 1; optimizationSettings->starting_points <= 256;
				optimizationSettings->starting_points = optimizationSettings->starting_points * 16) {
		optimizationSettings->batch_size = optimizationSettings->starting_points;
			mt->start();
			SPCASolver::GPUSolver::gpu_sparse_PCA_solver(handle, m, n, d_B, h_x, optimizationSettings,
					optimizationStatistics, LD_M, LD_N);
			mt->end();
			std::vector<F> x(n, 0);
			for (int i = 0; i < n; i++)
				x[i] = h_x[i];
			logTime(fileOut, mt, optimizationStatistics, optimizationSettings, x, m, n);


			// CPU
			mt->start();
			SPCASolver::MulticoreSolver::denseDataSolver(&h_B[0], LD_M, &x[0], m, n, optimizationSettings,
								optimizationStatistics);
			mt->end();
			logTime(fileOutCPU, mt, optimizationStatistics, optimizationSettings, x, m, n);

		}
	}
	fileOutCPU.close();
	fileOut.close();

	optimizationStatisticsus = hipblasDestroy(handle);
	if (optimizationStatisticsus != CUBLAS_optimizationStatisticsUS_SUCCESS) {
		fprintf(stderr, "!cublas shutdown error\n");
		return EXIT_FAILURE;
	}
	return 0;
}

int main(int argc, char *argv[]) {
	SolverStructures::OptimizationSettings* optimizationSettings =
			new OptimizationSettings();
	optimizationSettings->result_file = "results/gpu_unittest.txt";
	optimizationSettings->verbose = false;
	optimizationSettings->starting_points = 1024;
	optimizationSettings->batch_size = optimizationSettings->starting_points;
	optimizationSettings->onTheFlyMethod = false;
	optimizationSettings->gpu_use_k_selection_algorithm = false;
	optimizationSettings->constrain = 20;
	optimizationSettings->toll = 0.0001;
	optimizationSettings->max_it = 100;
	cout << "Double test" << endl;
	test_solver<double>(optimizationSettings);
	return 0;
}

