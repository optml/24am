/*
 *
 * This is a parallel sparse PCA solver
 *
 * The solver is based on a simple alternating maximization (AM) subroutine 
 * and is based on the paper
 *    P. Richtarik, M. Takac and S. Damla Ahipasaoglu 
 *    "Alternating Maximization: Unified Framework and 24 Parallel Codes for L1 and L2 based Sparse PCA"
 *
 * The code is available at https://code.google.com/p/24am/
 * under GNU GPL v3 License
 * 
 *    GPU SOLVER FOR SPARSE PCA - frontend console interface
 *
 */


#include "../class/optimization_settings.h"
#include "../class/optimization_statistics.h"
#include "../utils/file_reader.h"
#include "../utils/option_console_parser.h"
#include "../gpugpower/gpu_headers.h"


template<typename F>
int load_data_and_run_solver(SolverStructures::OptimizationSettings* optimizationSettings) {
	mytimer* mt = new mytimer();
	mt->start();
	SolverStructures::OptimizationStatisticsistics* optimizationStatistics =
			new OptimizationStatisticsistics();
	hipDeviceProp_t dp;
	hipGetDeviceProperties(&dp, 0);
	optimizationSettings->gpu_sm_count = dp.multiProcessorCount;
	optimizationSettings->gpu_max_threads = dp.maxThreadsPerBlock;

	unsigned int ldB;
	unsigned int m;
	unsigned int n;
	std::vector<F> B_mat;
	InputOuputHelper::readCSVFile(B_mat, ldB, m, n, optimizationSettings->data_file);
	optimizationStatistics->n = n;

	const int MEMORY_BANK_FLOAT_SIZE = MEMORY_ALIGNMENT / sizeof(F);
	const unsigned int LD_M = (
			m % MEMORY_BANK_FLOAT_SIZE == 0 ?
					m :
					(m / MEMORY_BANK_FLOAT_SIZE + 1) * MEMORY_BANK_FLOAT_SIZE);
	const unsigned int LD_N = (
			n % MEMORY_BANK_FLOAT_SIZE == 0 ?
					n :
					(n / MEMORY_BANK_FLOAT_SIZE + 1) * MEMORY_BANK_FLOAT_SIZE);
	thrust::host_vector<F> h_B(LD_M * n, 0);
	// get data into h_B;
	for (unsigned int row = 0; row < m; row++) {
		for (unsigned int col = 0; col < n; col++) {
			h_B[row + col * LD_M] = B_mat[row + col * m];
		}
	}
	// allocate vector for solution
	thrust::host_vector<F> h_x(n, 0);
	// move data to DEVICE
	thrust::device_vector<F> d_B = h_B;

	cublasoptimizationStatisticsus_t optimizationStatisticsus;
	hipblasHandle_t handle;
	optimizationStatisticsus = hipblasCreate(&handle);
	if (optimizationStatisticsus != CUBLAS_optimizationStatisticsUS_SUCCESS) {
		fprintf(stderr, "! CUBLAS initialization error\n");
		return EXIT_FAILURE;
	} else {
		printf("CUBLAS initialized.\n");
	}
//FIXME
	optimizationSettings->gpu_use_k_selection_algorithm = true;
	optimizationSettings->gpu_use_k_selection_algorithm = false;
	SPCASolver::GPUSolver::gpu_sparse_PCA_solver(handle, m, n, d_B, h_x, optimizationSettings, optimizationStatistics,
			LD_M, LD_N);
	mt->end();
	optimizationStatistics->total_elapsed_time = mt->getElapsedWallClockTime();
	InputOuputHelper::save_results(optimizationStatistics, optimizationSettings, &h_x[0], n);
	InputOuputHelper::save_optimizationStatisticsistics(optimizationStatistics, optimizationSettings);
	optimizationStatisticsus = hipblasDestroy(handle);
	if (optimizationStatisticsus != CUBLAS_optimizationStatisticsUS_SUCCESS) {
		fprintf(stderr, "!cublas shutdown error\n");
		return EXIT_FAILURE;
	}
	return 0;
}

int main(int argc, char *argv[]) {
	SolverStructures::OptimizationSettings* optimizationSettings =
			new OptimizationSettings();
	int optimizationStatisticsus = parseConsoleOptions(optimizationSettings, argc, argv);
	if (optimizationStatisticsus > 0)
		return optimizationStatisticsus;
	if (optimizationSettings->double_precission) {
		load_data_and_run_solver<double>(optimizationSettings);
	} else {
		load_data_and_run_solver<float>(optimizationSettings);
	}
	return 0;
}

